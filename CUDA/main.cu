#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include "readconfig.hpp"
#include "dataloader.hpp"

#define BLK_DIM 512



__global__ void forwardpass(
    float* layer_output,        //(dim: out_size)
    const float* layer_input,   //(dim: in_size)
    const float* weights,       //(dim: out_size x in_size)
    const float* biases,        //(dim: out_size)
    const int in_size,
    const int out_size
) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= out_size) {
        return;
    }

    float sum = 0.0f;
    for (int k = 0; k < in_size; k++) {
        sum += weights[i * in_size + k] * layer_input[k];
    }
    //Applying ReLU
    layer_output[i] = sum + biases[i] > 0 ? sum + biases[i] : 0.0f;
}


float* softmax(float* values, const int size) {
    // Launching a CUDA kernel for a 10-dimension vector would introduce more overhead than performances.
    // Hence, the softmax is not parallelized
    float max_val = values[0];
    for (int i = 1; i < size; i++) {
        if (values[i] > max_val) max_val = values[i];
    }
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        values[i] = expf(values[i] - max_val);
        sum += values[i];
    }
    for (int i = 0; i < size; i++) {
        values[i] /= sum;
    }
    return values;
}

void one_hot_encoding(int* encoded, int value, int size) {
    // This operation is not parallelized: the size needed is 10. Loading a cuda kernel would introduce
    // overhead
    for (int i=0; i<size; i++) {
        encoded[i] = 0;
    }
    encoded[value] = 1;
}

void backward_pass(float* layer, int size, const int* one_hot_label, const float* output_results) {
    // Same as the other utilities functions: since the layer size is small, launching a kernel will introduce
    // overhead
    for (int i=0; i < size; i++) {
        const float y = static_cast<float>(one_hot_label[i]);
        const float o = output_results[i];
        //simplified formula, output_results comes from a softmax. Otherwise Jacobians would be involved.
        layer[i] = o - y;
    }
}



__global__ void compute_hidden_delta(
    float* delta,
    const float* next_delta,
    const float* weights,
    const float* values,
    int layer_size,
    int next_size
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= layer_size) return;

    float sum = 0.0f;
    for (int j = 0; j < next_size; j++) {
        sum += weights[j * layer_size + i] * next_delta[j];
    }

    float val = values[i];
    float deriv = (val > 0) ? 1.0f : 0.0f; // ReLU derivative
    delta[i] = deriv * sum;
}


__global__ void update_weights(
    float* weights,
    const float* values,
    const float* next_delta,
    int layer_size,
    int next_size,
    float lr
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_weights = layer_size * next_size;
    if (idx >= total_weights) return;

    int i = idx / next_size;
    int j = idx % next_size;


    weights[j * layer_size + i] -= lr * values[i] * next_delta[j];
}


__global__ void update_biases(
    float* biases,
    const float* next_delta,
    int next_size,
    float lr
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= next_size) return;
    biases[i] -= lr * next_delta[i];
}


int main() {
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distribution(-0.5, 0.5);
    int train_size, test_size, epochs;
    int layer_sizes[MAX_LAYERS];
    int size;

    printf("Reading configurations... \n");
    read_config("../config.txt", &train_size, &test_size, &epochs, layer_sizes, &size);
    printf("Done!\n");
    printf("Network has %i layers\n",size);
    printf("[ ");

    int total_nodes = 0;
    int total_biases = 0;
    int total_weights = 0;

    int* h_bias_indexes = new int[size-1];
    int* h_weights_indexes = new int[size-1];
    int* h_values_indexes = new int[size];
    int* h_deltas_indexes = new int[size];

    h_bias_indexes[0] = 0;
    h_weights_indexes[0] = 0;
    h_values_indexes[0] = 0;
    h_deltas_indexes[0] = 0;

    // Filling the indexes arrays
    for (int i=1; i<size-1; i++) {
        h_weights_indexes[i] = layer_sizes[i-1] * layer_sizes[i] + h_weights_indexes[i-1];
        h_bias_indexes[i] = layer_sizes[i] + h_bias_indexes[i-1];
    }

    for (int i=1; i<size; i++) {
        h_values_indexes[i] = layer_sizes[i-1] + h_values_indexes[i-1];
        h_deltas_indexes[i] = layer_sizes[i-1] + h_deltas_indexes[i-1];
    }

    for (int i = 0; i < size; i++) {
        total_nodes += layer_sizes[i];
        if (i != 0) total_biases += layer_sizes[i];
        if (i < size - 1) {
            total_weights += layer_sizes[i] * layer_sizes[i+1];
        }
        printf("%i ", layer_sizes[i]);
    }
    printf("] \n");


    double network_size_kb = static_cast<double>(total_weights) * sizeof(float) / 1024.0;
    double dataset_size_mb = (static_cast<double>(train_size) + static_cast<double>(test_size)) * IMAGE_SIZE * sizeof(float) / (1024.0 * 1024.0);
    printf("Network has %i total weights (%.2f KB) and %i total nodes\n", total_weights, network_size_kb, total_nodes);
    printf("The dataset will occupy %.2f MB on the GPU\n", dataset_size_mb);

    float* h_network_values = new float[total_nodes];
    float* h_network_deltas = new float[total_nodes];
    float* h_network_biases = new float[total_biases];
    float* h_network_weights = new float[total_weights];
    int* h_onehot_label = new int[layer_sizes[size-1]];

    // initialize host arrays
    for (int i=0; i < total_nodes; i++) {
        h_network_values[i] = 0.0f;
        h_network_deltas[i] = 0.0f;
    }

    for (int i=0; i < total_biases; i++) {
        h_network_biases[i] = 0.1f;
    }

    for (int i=0; i < total_weights; i++) {
        h_network_weights[i] = distribution(generator);
    }

    //Creating pointers to device memory
    float* d_network_values;
    float* d_network_deltas;
    float* d_network_biases;
    float* d_network_weights;


    // hipMalloc
    hipMalloc(&d_network_values, total_nodes * sizeof(float));
    hipMalloc(&d_network_deltas, total_nodes * sizeof(float));
    hipMalloc(&d_network_biases, total_biases * sizeof(float));
    hipMalloc(&d_network_weights, total_weights * sizeof(float));

    // copying from host
    hipMemcpy(d_network_values, h_network_values, total_nodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_network_deltas, h_network_deltas, total_nodes * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_network_biases, h_network_biases, total_biases * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_network_weights, h_network_weights, total_weights * sizeof(float), hipMemcpyHostToDevice);


    // Loading the dataset
    float *X_train, *X_test;
    int *Y_train, *Y_test;
    read_dataset(&X_train, &Y_train, true);
    read_dataset(&X_test, &Y_test, false);

    // Creating pointers to device memory
    float* d_X_train;

    //hipMalloc
    hipMalloc(&d_X_train, train_size * IMAGE_SIZE * sizeof(float));

    //hipMemcpy
    hipMemcpy(d_X_train, X_train, train_size * IMAGE_SIZE * sizeof(float), hipMemcpyHostToDevice);

    for (int epoch = 0; epoch < epochs; epoch++) {
        printf("Epoch %d/%d\n", epoch + 1, epochs);
        // For each epoch, cycle on every image
        int correct = 0;
        for (int image_index = 0; image_index < train_size; image_index++) {
            float* d_input_image = d_X_train + image_index * IMAGE_SIZE;
            hipMemcpy(d_network_values, d_input_image, layer_sizes[0] * sizeof(float), hipMemcpyDeviceToDevice);

            for (int layer_index = 0; layer_index < size-1; layer_index++) {
                int in_size = layer_sizes[layer_index];
                int out_size = layer_sizes[layer_index+1];

                float* d_layer_input = d_network_values + h_values_indexes[layer_index];
                float* d_layer_output = d_network_values + h_values_indexes[layer_index+1];
                float* d_layer_weights = d_network_weights + h_weights_indexes[layer_index];
                // May seems like an error but it isn't: h_deltas_indexes stores indexes already shifted, so
                // layer_index is ok, layer_index wouldn't
                float* d_layer_biases = d_network_biases + h_bias_indexes[layer_index];

                int blocks = (out_size + BLK_DIM - 1) / BLK_DIM;
                forwardpass<<<blocks, BLK_DIM>>>(
                    d_layer_output,
                    d_layer_input,
                    d_layer_weights,
                    d_layer_biases,
                    in_size,
                    out_size
                );
                hipDeviceSynchronize();


                }

                // Compute training accuracy
                float* h_network_output = new float[layer_sizes[size-1]];
                float* h_network_deltas_last_layer = new float[layer_sizes[size-1]];

                float* d_network_output = d_network_values + h_values_indexes[size-1];
                hipMemcpy(h_network_output, d_network_output, layer_sizes[size-1] * sizeof(float), hipMemcpyDeviceToHost);
                h_network_output = softmax(h_network_output, layer_sizes[size-1]);
                int idx_max = 0;
                for (int i=0; i < layer_sizes[size-1]; i++) {
                    if (h_network_output[i] > h_network_output[idx_max]) idx_max = i;
                }
                if (Y_train[image_index] == idx_max) correct++;

                //#1 Compute loss value
                //      one hot encoding
                one_hot_encoding(h_onehot_label,Y_train[image_index],layer_sizes[size-1]);


                //#2 Backward pass
                backward_pass(h_network_deltas_last_layer, layer_sizes[size-1], h_onehot_label, h_network_output);

                // Now I need to copy everything on device again to compute the forward pass
                hipMemcpy(d_network_deltas + h_deltas_indexes[size-1],
                   h_network_deltas_last_layer,
                   layer_sizes[size-1] * sizeof(float),
                   hipMemcpyHostToDevice);

                //#3 Backpropagation
                for (int l = size-2; l >= 0; l--) {
                    int layer_size = layer_sizes[l];
                    int next_size  = layer_sizes[l+1];

                    float* d_layer_values  = d_network_values + h_values_indexes[l];
                    float* d_layer_delta   = d_network_deltas + h_deltas_indexes[l];
                    float* d_next_delta    = d_network_deltas + h_deltas_indexes[l+1];
                    float* d_layer_weights = d_network_weights + h_weights_indexes[l];
                    float* d_layer_biases  = d_network_biases + h_bias_indexes[l];
                    int blocks = (layer_size + BLK_DIM - 1) / BLK_DIM;  // per compute_hidden_delta

                    // --- Compute delta ---
                    compute_hidden_delta<<<blocks, BLK_DIM>>>(
                        d_layer_delta, d_next_delta, d_layer_weights, d_layer_values,
                        layer_size, next_size
                    );
                    hipDeviceSynchronize();

                    float learning_rate = 0.01f;
                    // --- Update weights ---
                    int total_weights_layer = layer_size * next_size;

                    blocks = (total_weights_layer + BLK_DIM - 1) / BLK_DIM;
                    update_weights<<<blocks, BLK_DIM>>>(
                        d_layer_weights, d_layer_values, d_next_delta,
                        layer_size, next_size, learning_rate
                    );

                    // --- Update biases ---
                    blocks = (next_size + BLK_DIM - 1) / BLK_DIM;
                    update_biases<<<blocks, BLK_DIM>>>(d_layer_biases, d_next_delta, next_size, learning_rate);
                    hipDeviceSynchronize();


            }

        }
        // Retrieve the last layer values: they should be equal for each epoch (for each run)
        printf("Accuracy: %f\n", static_cast<float>(correct) / static_cast<float>(train_size));

    }






    return 0;
}
